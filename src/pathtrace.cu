#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/partition.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"

#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtc/matrix_inverse.hpp>

#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#include <>
#include "cfg.h"

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)


#pragma region feature_parameter
//# define camera_jittering 1 // camera antialiasing

#pragma endregion
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
    getchar();
#  endif
    exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
        int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene * hst_scene = NULL;
static glm::vec3 * dev_image = NULL;
static Geom * dev_geoms = NULL;
static int* dev_lightIDs = NULL;
// add triangles
static Triangle* dev_triangles = NULL;
static GLTF_Model* dev_gltf_models = NULL;

static Material * dev_materials = NULL;
static PathSegment * dev_paths = NULL;
static ShadeableIntersection * dev_intersections = NULL;

static glm::vec3* dev_textures = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...

#if cache_first_bounce
static ShadeableIntersection* dev_first_intersections_cache = NULL;
#endif

#if material_sort_ID
static int * dev_materialIDs = NULL;
thrust::device_ptr<ShadeableIntersection> dev_thrust_intersections;
thrust::device_ptr<PathSegment> dev_thrust_paths;
thrust::device_ptr<int> dev_thrust_ID;
thrust::zip_iterator<
    thrust::tuple<
    thrust::device_ptr<ShadeableIntersection>,
    thrust::device_ptr<PathSegment>
    >
> zip_it;
#endif

void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

  	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

  	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
  	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);
    // light IDs
    hipMalloc(&dev_lightIDs, scene->lightIDs.size() * sizeof(int));
    hipMemcpy(dev_lightIDs, scene->lightIDs.data(), scene->lightIDs.size() * sizeof(int), hipMemcpyHostToDevice);
    // add triangles
    hipMalloc(&dev_triangles, scene->triangles.size() * sizeof(Triangle));
    hipMemcpy(dev_triangles, scene->triangles.data(), scene->triangles.size() * sizeof(Triangle), hipMemcpyHostToDevice);
    // and gltf models
    hipMalloc(&dev_gltf_models, scene->gltf_models.size() * sizeof(GLTF_Model));
    hipMemcpy(dev_gltf_models, scene->gltf_models.data(), scene->gltf_models.size() * sizeof(GLTF_Model), hipMemcpyHostToDevice);

  	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
  	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

  	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
  	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    // TODO: initialize any extra device memeory you need
#if cache_first_bounce
    hipMalloc(&dev_first_intersections_cache, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_first_intersections_cache, 0, pixelcount * sizeof(ShadeableIntersection));
#endif

#if material_sort_ID
    hipMalloc(&dev_materialIDs, pixelcount * sizeof(int));
    hipMemset(dev_materialIDs, 0,  pixelcount * sizeof(int));
#endif
    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    hipFree(dev_image);  // no-op if dev_image is null
  	hipFree(dev_paths);
  	hipFree(dev_geoms);
    hipFree(dev_lightIDs);
  	hipFree(dev_materials);
  	hipFree(dev_intersections);

    hipFree(dev_triangles);
    hipFree(dev_gltf_models);
    // TODO: clean up any extra device memory you created
#if cache_first_bounce
    hipFree(dev_first_intersections_cache);
#endif
#if material_sort
#if material_sort_ID
    hipFree(dev_materialIDs);
#endif
#endif
    checkCUDAError("pathtraceFree");
}


__host__ __device__
glm::vec2 ConcentricSampleDisk(
    thrust::default_random_engine& rng
    ) {
    // pbrt 
    thrust::uniform_real_distribution<float> u01(0, 1);
    glm::vec2 t(u01(rng), u01(rng));
    // map to [-1, -1]
    t = 2.0f * t - glm::vec2(1.0f, 1.0f);
    if (t.x == 0.0f && t.y == 0.0f) {
        return t;
    }
    float theta, r;
    if (abs(t.x) > abs(t.y)) {
        r = t.x;
        theta = (PI / 4.0f) * t.y / t.x;
    }
    else {
        r = t.y;
        theta = (PI / 2.0f) - PI / 4.0f * t.x / t.y;
    }

    return r * glm::vec2(cos(theta), sin(theta));
    
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment & segment = pathSegments[index];

		segment.ray.origin = cam.position;
        segment.colorSum = glm::vec3(0.);
        segment.colorThroughput = glm::vec3(1.);
        
		// TODO: implement antialiasing by jittering the ray
        thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
        thrust::uniform_real_distribution<float> du(0.0, 0.5);
        segment.ray.time = du(rng);
#if camera_jittering
        
        //thrust::uniform_real_distribution<float> u01(0, 1);
        thrust::uniform_real_distribution<float> u01(-0.5, 0.5);
		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x + u01(rng) - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y + u01(rng) - (float)cam.resolution.y * 0.5f)
			);

#else
        segment.ray.direction = glm::normalize(
            cam.view
            - cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
            - cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
        );
#endif

#if dof
        // pbrt 6.2.3
        //thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
        glm::vec2 offset = ConcentricSampleDisk(rng) * cam.apertureRadius;

        float ft = abs(cam.focusDist / segment.ray.direction.z);
        //glm::vec3 pFocus = getPointOnRay(segment.ray, ft);
        glm::vec3 pFocus = ft * segment.ray.direction;

        segment.ray.origin += glm::vec3(offset, 0.0f);
        segment.ray.direction = glm::normalize(pFocus - glm::vec3(offset, 0.0f));
#endif

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

__host__ __device__
glm::mat4 dev_buildTransformationMatrix(glm::vec3 translation, glm::vec3 rotation, glm::vec3 scale) {
    glm::mat4 translationMat = glm::translate(glm::mat4(), translation);
    glm::mat4 rotationMat = glm::rotate(glm::mat4(), rotation.x * (float)PI / 180, glm::vec3(1, 0, 0));
    rotationMat = rotationMat * glm::rotate(glm::mat4(), rotation.y * (float)PI / 180, glm::vec3(0, 1, 0));
    rotationMat = rotationMat * glm::rotate(glm::mat4(), rotation.z * (float)PI / 180, glm::vec3(0, 0, 1));
    glm::mat4 scaleMat = glm::scale(glm::mat4(), scale);
    return translationMat * rotationMat * scaleMat;
}
// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment * pathSegments
	, Geom * geoms
	, int geoms_size
	, ShadeableIntersection * intersections,
    GLTF_Model* models = dev_gltf_models,
    Triangle* triangles = dev_triangles
	)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

        SceneIntersection(pathSegment.ray, geoms, geoms_size, models, triangles, intersections[path_index]);
	}
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial (
  int iter
  , int num_paths
	, ShadeableIntersection * shadeableIntersections
	, PathSegment * pathSegments
	, Material * materials
	)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_paths)
  {
    ShadeableIntersection intersection = shadeableIntersections[idx];
    if (intersection.t > 0.0f) { // if the intersection exists...
      // Set up the RNG
      // LOOK: this is how you use thrust's RNG! Please look at
      // makeSeededRandomEngine as well.
      thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
      thrust::uniform_real_distribution<float> u01(0, 1);

      Material material = materials[intersection.materialId];
      glm::vec3 materialColor = material.color;

      // If the material indicates that the object was a light, "light" the ray
      if (material.emittance > 0.0f) {
        pathSegments[idx].colorSum *= (materialColor * material.emittance);
      }
      // Otherwise, do some pseudo-lighting computation. This is actually more
      // like what you would expect from shading in a rasterizer like OpenGL.
      // TODO: replace this! you should be able to start with basically a one-liner
      else {
        float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(0.0f, 1.0f, 0.0f));
        pathSegments[idx].colorSum *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
        pathSegments[idx].colorSum *= u01(rng); // apply some noise because why not
      }
    // If there was no intersection, color the ray black.
    // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
    // used for opacity, in which case they can indicate "no opacity".
    // This can be useful for post-processing and image compositing.
    } else {
      pathSegments[idx].colorSum = glm::vec3(0.0f);
    }
  }
}

#pragma region myMaterial
__global__ void shadeTrueMaterial(
    int iter,
    int max_depth,
    int num_paths,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    int *lightIDs,
    int light_size,
    Geom* geoms,
    int geom_size,
    Material* materials,
    Triangle* triangles,
    GLTF_Model* gltf_models,
    glm::vec3* textures,
    glm::vec3* dev_image
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_paths)
    {
        ShadeableIntersection intersection = shadeableIntersections[idx];
        PathSegment& cur_pathSegment = pathSegments[idx];
        
        if (intersection.t > 0.0f) { // if the intersection exists...
            // Set up the RNG
            // LOOK: this is how you use thrust's RNG! Please look at
            // makeSeededRandomEngine as well.
            thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
            thrust::uniform_real_distribution<float> u01(0, 1);

            Material material = materials[intersection.materialId];
            glm::vec3 materialColor = material.color;


            // If the material indicates that the object was a light, "light" the ray
            if (material.emittance > 0.0f) {
#if DirectLightPass == 1:
                bool specularBounce = true; // TODO make this really specular bounce
                if (cur_pathSegment.remainingBounces == max_depth || specularBounce) {
                    cur_pathSegment.colorSum += cur_pathSegment.colorThroughput * (materialColor * material.emittance);
                    // stop if hit a light
                    cur_pathSegment.remainingBounces = 0;
                }
#else
                cur_pathSegment.colorSum += cur_pathSegment.colorThroughput * (materialColor * material.emittance);
                // stop if hit a light
                cur_pathSegment.remainingBounces = 0;
#endif
            }
            // Otherwise, do some pseudo-lighting computation. This is actually more
            // like what you would expect from shading in a rasterizer like OpenGL.
            // TODO: replace this! you should be able to start with basically a one-liner
            else {
                // TODO normal mapping
                glm::vec3 n = intersection.surfaceNormal;

#if DirectLightPass == 1
                UniformSampleOneLight(
                    cur_pathSegment,
                    intersection,
                    materials,
                    -cur_pathSegment.ray.direction,
                    light_size,
                    lightIDs,
                    geoms,
                    geom_size,
                    gltf_models,
                    triangles,
                    textures,
                    rng
                );

#endif
#if InDirectLightPass == 1
                scatterRay(
                    cur_pathSegment,
                    intersection,
                    material,
                    textures,
                    rng
                );
#else:
                cur_pathSegment.remainingBounces = 0;
#endif
                

            }
            
        }
        else {
            // If there was no intersection, color the ray black.
            // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
            // used for opacity, in which case they can indicate "no opacity".
            // This can be useful for post-processing and image compositing.
            cur_pathSegment.colorThroughput = glm::vec3(0.0f);
            cur_pathSegment.remainingBounces = 0;
        }
    }
        
    
}
#pragma endregion

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 * image, PathSegment * iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.colorSum;
	}
}


// ref:https://thrust.github.io/doc/group__stream__compaction_ga5fa8f86717696de88ab484410b43829b.html
struct parition_not_end
{
    __host__ __device__
        bool operator()(const PathSegment& ps)
    {
        return ps.remainingBounces > 0;
    }
};

struct material_operator_bigger {
    __host__ __device__
        bool operator()(const ShadeableIntersection& intsct1, const ShadeableIntersection& intsct2)
    {

        return intsct1.materialId > intsct2.materialId;
    }
};


struct materialID_operator_bigger {
    __host__ __device__
        bool operator()(const int& id1, const int& id2)
    {

        return id1 > id2;
    }
};



/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4 *pbo, int frame, int iter) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * TODO: Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

    // TODO: perform one iteration of path tracing

	generateRayFromCamera <<<blocksPerGrid2d, blockSize2d >>>(cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

    bool iterationComplete = false;
	while (!iterationComplete) {

        // clean shading chunks
        hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

        // tracing
        dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
#if cache_first_bounce
        if (iter == 1 && depth == 0) {
            computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
                depth
                , num_paths
                , dev_paths
                , dev_geoms
                , hst_scene->geoms.size()
                , dev_first_intersections_cache
                );
            checkCUDAError("no caching first bounce trace one bounce");
            //hipDeviceSynchronize();
            hipMemcpy(dev_intersections, dev_first_intersections_cache, pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
        }
        else {
            if (depth == 0) {
                hipMemcpy( dev_intersections, dev_first_intersections_cache, pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
            }
            else {
                // can not cache the rest depth bounce
                computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
                    depth
                    , num_paths
                    , dev_paths
                    , dev_geoms
                    , hst_scene->geoms.size()
                    , dev_intersections
                    );
                checkCUDAError("no caching first bounce trace one bounce");
                //hipDeviceSynchronize();
            }
        }
#else
        computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
            depth
            , num_paths
            , dev_paths
            , dev_geoms
            , hst_scene->geoms.size()
            , dev_intersections
            );
        checkCUDAError("no caching first bounce trace one bounce");
        //hipDeviceSynchronize();
#endif // cache_first_bounce

        
       
        depth++;


        // TODO:
        // --- Shading Stage ---
        // Shade path segments based on intersections and generate new rays by
        // evaluating the BSDF.
        // Start off with just a big kernel that handles all the different
        // materials you have in the scenefile.
        // TODO: compare between directly shading the path segments and shading
        // path segments that have been reshuffled to be contiguous in memory.
#if material_sort
#if material_sort_ID
         // map intersections -> ID(int) to trigger radix sort
        construct_materialIDs << <numblocksPathSegmentTracing, blockSize1d >> > (num_paths, dev_intersections, dev_materialIDs);
        // have to sort both intersections and paths by key
        // ref https://stackoverflow.com/questions/6617066/sorting-3-arrays-by-key-in-cuda-using-thrust-perhaps/42484689#42484689
        /*
        thrust::device_vector<ShadeableIntersection> dev_thrust_intersections_vec(dev_intersections, dev_intersections + num_paths);
        thrust::device_vector<PathSegment> dev_thrust_paths_vec(dev_paths, dev_paths + num_paths);
        thrust::device_vector<int> dev_thrust_ID_vec(dev_materialIDs, dev_materialIDs + num_paths);
        thrust::zip_iterator<
            thrust::tuple<
                thrust::device_vector<ShadeableIntersection>::iterator, 
                thrust::device_vector<PathSegment>::iterator>
            > 
            zip_it 
            = thrust::make_zip_iterator(thrust::make_tuple(dev_thrust_intersections_vec.begin(), dev_thrust_paths_vec.begin()));
        thrust::sort_by_key(dev_thrust_ID_vec.begin(), dev_thrust_ID_vec.end(), zip_it);
        */
        dev_thrust_intersections = thrust::device_ptr<ShadeableIntersection>(dev_intersections);
        dev_thrust_paths = thrust::device_ptr<PathSegment>(dev_paths);
        dev_thrust_ID = thrust::device_ptr<int>(dev_materialIDs);
        zip_it = thrust::make_zip_iterator(thrust::make_tuple(dev_thrust_intersections, dev_thrust_paths));
        thrust::sort_by_key(dev_thrust_ID, dev_thrust_ID + num_paths, zip_it);

#else
        thrust::sort_by_key(thrust::device, dev_intersections, dev_intersections + num_paths, dev_paths, material_operator_bigger());

#endif
       
        #endif
        shadeTrueMaterial<<<numblocksPathSegmentTracing, blockSize1d>>> (
        iter,
        traceDepth,
        num_paths,
        dev_intersections,
        dev_paths,
        dev_lightIDs,
        hst_scene->lightIDs.size(),
        dev_geoms,
        hst_scene->geoms.size(),
        dev_materials,
        dev_triangles,
        dev_gltf_models,
        dev_textures,
        dev_image
        );
        if (depth > traceDepth) {
            iterationComplete = true;
        }

         // Done: end based off stream compaction results.
        dev_path_end = thrust::stable_partition(thrust::device, dev_paths, dev_paths + num_paths, parition_not_end());
        if (dev_path_end == dev_paths) {
            iterationComplete = true;
        }
        else {
            num_paths = dev_path_end - dev_paths;
        }


	}

  // Assemble this iteration and apply it to the image
    
	//finalGather<<<numBlocksPixels, blockSize1d>>>(num_paths, dev_image, dev_paths);
    finalGather << <numBlocksPixels, blockSize1d >> > (pixelcount, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
            pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}

__global__ void correctTexturesKernel(glm::vec3* texture, glm::vec3 gamma, int size)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < size)
        texture[index] = glm::pow(texture[index], gamma);
}

void initDeviceTexture(Scene* scene)
{
    int totalMemory = 0;

    for (int i = 0; i < scene->textures.size(); i++)
        totalMemory += scene->textures[i]->xSize * scene->textures[i]->ySize;

    std::cout << "Total texture memory: " << totalMemory << std::endl;

    std::vector<int> offsetList;

    if (totalMemory > 0)
    {
        hipMalloc(&dev_textures, totalMemory * sizeof(glm::vec3));

        const int blockSize1d = 128;

        int offset = 0;
        for (int i = 0; i < scene->textures.size(); i++)
        {
            offsetList.push_back(offset);

            Texture* tex = scene->textures[i];
            int size = tex->xSize * tex->ySize;
            hipMemcpy(dev_textures + offset, tex->pixels, size * sizeof(glm::vec3), hipMemcpyHostToDevice);

            glm::vec3 gamma = glm::vec3(tex->gamma);
            dim3 numBlocksPixels = (size + blockSize1d - 1) / blockSize1d;
            correctTexturesKernel << <numBlocksPixels, blockSize1d >> > (dev_textures + offset, gamma, size);

            offset += size;
        }
    }

    // Now we need to set all texture descriptor indices
    /*if (scene->state.camera.bokehTexture.index >= 0)
        scene->state.camera.bokehTexture.index = offsetList[scene->state.camera.bokehTexture.index];*/

    for (Material& m : scene->materials)
    {
        if (m.diffuseTexture.index >= 0)
            m.diffuseTexture.index = offsetList[m.diffuseTexture.index];

        if (m.specularTexture.index >= 0)
            m.specularTexture.index = offsetList[m.specularTexture.index];

        if (m.normalTexture.index >= 0)
            m.normalTexture.index = offsetList[m.normalTexture.index];
    }

    checkCUDAError("initializeDeviceTextures");
}
