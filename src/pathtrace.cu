#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/partition.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "glm/gtx/transform.hpp"
#include "glm/gtx/rotate_vector.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1
#define USE_SHADE_MATERIAL
#define SORT_RAYS_BY_MATERIALS 
//#define CACHE_FIRST_BOUNCE 
#define STREAM_COMPACT_RAYS 
#define ANTI_ALIASING
//#define DEPTH_OF_FIELD
#define DIRECT_LIGHTING 
//#define MOTION_BLUR 
//#define MOTION_BLUR_2 //Ghost mode lol 
//#define BOKEH
#define BOUNDING_VOLUME

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line) {
#if ERRORCHECK
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (hipSuccess == err) {
		return;
	}

	fprintf(stderr, "CUDA error");
	if (file) {
		fprintf(stderr, " (%s:%d)", file, line);
	}
	fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
	getchar();
#  endif
	exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
	int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
	int iter, glm::vec3* image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);
		glm::vec3 pix = image[index];

		glm::ivec3 color;
		color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
		color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
		color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = color.x;
		pbo[index].y = color.y;
		pbo[index].z = color.z;
	}
}

static Scene* hst_scene = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...
static ShadeableIntersection* dev_cache_intersections = NULL;
//Trying 2 different approaches - change after finalizing 
static Triangle* dev_mesh_triangles = NULL;
//static Mesh* dev_mesh = 0; 
static int* dev_num_triangles = 0;
static Geom* dev_lights = 0;

void pathtraceInit(Scene* scene) {
	hst_scene = scene;
	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	// TODO: initialize any extra device memeory you need
	hipMalloc(&dev_cache_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_cache_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	//Copy Mesh Data from host to device 
	//hipMalloc(&dev_mesh, sizeof(Mesh));
	//hipMemcpy(dev_mesh, &scene->mesh, sizeof(Mesh), hipMemcpyHostToDevice);

	hipMalloc(&dev_mesh_triangles, scene->mesh.triangles.size() * sizeof(Triangle));
	hipMemcpy(dev_mesh_triangles, scene->mesh.triangles.data(), scene->mesh.triangles.size() * sizeof(Triangle), hipMemcpyHostToDevice);

	hipMalloc(&dev_num_triangles, sizeof(int));
	hipMemcpy(dev_num_triangles, &scene->mesh.num_triangles, sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&dev_lights, scene->lights.size() * sizeof(Geom));
	hipMemcpy(dev_lights, scene->lights.data(), scene->lights.size() * sizeof(Geom), hipMemcpyHostToDevice);

	checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
	hipFree(dev_image);  // no-op if dev_image is null
	hipFree(dev_paths);
	hipFree(dev_geoms);
	hipFree(dev_materials);
	hipFree(dev_intersections);
	// TODO: clean up any extra device memory you created
	hipFree(dev_cache_intersections);
	//hipFree(dev_mesh);
	hipFree(dev_mesh_triangles);
	hipFree(dev_num_triangles);
	hipFree(dev_lights);
	checkCUDAError("pathtraceFree");
}

__host__ __device__ glm::vec3 squareToDiskConcentric(const glm::vec2& sample)
{
	glm::vec2 uOffset = 2.f * sample - glm::vec2(1.f, 1.f);
	if (uOffset.x == 0.f && uOffset.y == 0.f)
	{
		return glm::vec3(0.f, 0.f, 0.f);
	}
	float theta, r;
	if (std::abs(uOffset.x) > std::abs(uOffset.y))
	{
		r = uOffset.x;
		theta = (PI / 4) * (uOffset.y / uOffset.x);
	}
	else
	{
		r = uOffset.y;
		theta = (PI / 2) - (PI / 4) * (uOffset.x / uOffset.y);
	}
	return r * glm::vec3(std::cos(theta), std::sin(theta), 0.f);
}


__host__ __device__ float getSquared(float x) { return x * x; }

__host__ __device__ glm::vec3 squareToBokeh(const glm::vec2& sample)
{
	//Rejection Sampling
	//Get point on Disc
	glm::vec3 p = glm::vec3(sample, 0.f);
	p = p * 2.f - glm::vec3(1.f);
	p *= 1.5f;
	float x = p.x, y = p.y;
	//Reject point if it doesn't lie on heart
	if ((getSquared(x) + getSquared((5 * y / 4.f) - sqrt(abs(x)))) - 1 < 0) {
		return p;
	}
	return glm::vec3(0.f);
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) {
		int index = x + (y * cam.resolution.x);
		PathSegment& segment = pathSegments[index];

		segment.ray.origin = cam.position;
		thrust::default_random_engine rng = makeSeededRandomEngine(iter, x, 0);
		thrust::uniform_real_distribution<float> u01(0, 1);

#ifdef MOTION_BLUR_2 

		//Jitter the ray randomly about any axes 
		glm::vec3 jitteredRayOrigin = u01(rng) * glm::vec3(0.5f, 1.25f, 0.f);
		segment.ray.origin += jitteredRayOrigin;
#endif // MOTION_BLUR_2 

		segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

		// TODO: implement antialiasing by jittering the ray
#ifdef ANTI_ALIASING && !CACHE_FIRST_BOUNCE
		float x_offset = u01(rng), y_offset = u01(rng);

		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f + x_offset)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f + y_offset)
		);
#else
		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
		);
#endif // ANTI_ALIASING

#ifdef DEPTH_OF_FIELD
		if (cam.lensRadius > 0.f)
		{
			//Refer to 561 Path Tracer 
			glm::vec2 sample = glm::vec2(u01(rng), u01(rng));
			glm::vec2 point_on_lens(0.f);

#ifdef BOKEH
			point_on_lens = glm::vec2(cam.lensRadius * glm::vec3(glm::rotate(sample, 45.f), 0.f));
#else
			point_on_lens = glm::vec2(squareToDiskConcentric(sample)) * cam.lensRadius;
#endif // BOKEH
			glm::vec3 pof = cam.position + (cam.focalDistance * segment.ray.direction);

			segment.ray.origin = cam.position + (cam.up * point_on_lens.y) + (cam.right * point_on_lens.x);
			segment.ray.direction = glm::normalize(pof - segment.ray.origin);
		}
#endif // DEPTH_OF_FIELD
		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
	int depth
	, int num_paths
	, PathSegment* pathSegments
	, Geom* geoms
	, int geoms_size
	, ShadeableIntersection* intersections
	, Triangle* triangles
	, int num_triangles
	, int iter
)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		PathSegment pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms

		for (int i = 0; i < geoms_size; i++)
		{
			Geom& geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
#ifdef MOTION_BLUR
				thrust::default_random_engine rng = makeSeededRandomEngine(iter, path_index, 0);
				thrust::uniform_real_distribution<float> u01(0, 1);
				//Jitter the ray randomly about any axes 
				Ray jittered = pathSegment.ray; 
				jittered.origin += u01(rng) * glm::vec3(0.25f, 0.75f, 0.f);
				t = sphereIntersectionTest(geom, jittered, tmp_intersect, tmp_normal, outside);
#else
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
#endif //MOTION_BLUR
			}
			// TODO: add more intersection tests here... triangle? metaball? CSG?
			else if (geom.type == MESH)
			{
#ifdef BOUNDING_VOLUME
				bool success = false;
				meshBoundingVolumeTest(geom, pathSegment.ray, geom.geomMinCorner, geom.geomMinCorner, tmp_intersect, success); 
				if (success)
				{
					t = trianglesIntersectionTest(geom, triangles, num_triangles, pathSegment.ray, tmp_intersect, tmp_normal, outside);
				}
#else
				//t = meshIntersectionTest(mesh, pathSegment.ray, tmp_intersect, tmp_normal, outside);
				t = trianglesIntersectionTest(geom, triangles, num_triangles, pathSegment.ray, tmp_intersect, tmp_normal, outside);

#endif //BOUNDING_VOLUME

			}

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;
		}
	}
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial(
	int iter
	, int num_paths
	, ShadeableIntersection* shadeableIntersections
	, PathSegment* pathSegments
	, Material* materials
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) { // if the intersection exists...
		  // Set up the RNG
		  // LOOK: this is how you use thrust's RNG! Please look at
		  // makeSeededRandomEngine as well.
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
			thrust::uniform_real_distribution<float> u01(0, 1);

			Material material = materials[intersection.materialId];
			glm::vec3 materialColor = material.color;

			// If the material indicates that the object was a light, "light" the ray
			if (material.emittance > 0.0f) {
				pathSegments[idx].color *= (materialColor * material.emittance);
			}
			// Otherwise, do some pseudo-lighting computation. This is actually more
			// like what you would expect from shading in a rasterizer like OpenGL.
			// TODO: replace this! you should be able to start with basically a one-liner
			else {
				float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(0.0f, 1.0f, 0.0f));
				pathSegments[idx].color *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
				pathSegments[idx].color *= u01(rng); // apply some noise because why not
			}
			// If there was no intersection, color the ray black.
			// Lots of renderers use 4 channel color, RGBA, where A = alpha, often
			// used for opacity, in which case they can indicate "no opacity".
			// This can be useful for post-processing and image compositing.
		}
		else {
			pathSegments[idx].color = glm::vec3(0.0f);
		}
	}
}

//This is called in the shading stage 
__global__ void shadeMaterial(
	int iter
	, int num_paths
	, ShadeableIntersection* shadeableIntersections
	, PathSegment* pathSegments
	, Material* materials
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		ShadeableIntersection& currisect = shadeableIntersections[idx];
		PathSegment& currpath = pathSegments[idx];
		if (currpath.remainingBounces > 0 && currisect.t > 0.f) { // if the intersection still exists...

			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
			thrust::uniform_real_distribution<float> u01(0, 1);

			Material material = materials[currisect.materialId];
			glm::vec3 materialColor = material.color;

			// If the material indicates that the object was a light, "light" the ray
			// If object is light, stop bouncing
			if (material.emittance > 0.0f) {
				currpath.remainingBounces = 0;
				currpath.color *= (materialColor * material.emittance);
			}
			// Otherwise, do some pseudo-lighting computation. This is actually more
			// like what you would expect from shading in a rasterizer like OpenGL.
			else {
				glm::vec3 isect_normal = currisect.surfaceNormal;
				if (currpath.remainingBounces != 0)
				{
					scatterRay(currpath,
						getPointOnRay(currpath.ray, currisect.t),
						isect_normal,
						material,
						rng);
				}
				else
				{
					currpath.color = glm::vec3(0.f);
				}

			}
			--currpath.remainingBounces;
			// If there was no intersection, color the ray black.
			// Lots of renderers use 4 channel color, RGBA, where A = alpha, often
			// used for opacity, in which case they can indicate "no opacity".
			// This can be useful for post-processing and image compositing.
		}
		else {
			currpath.remainingBounces = 0;
			currpath.color = glm::vec3(0.0f);
		}
	}
}

__device__ __host__ int getRandLightIdx(int nLights, thrust::default_random_engine& rng)
{
	thrust::uniform_real_distribution<float> u01(0, 1);
	float rand = u01(rng);

	if (nLights == 0) return 0;
	int lightNum = glm::min((int)glm::floor(rand * nLights), nLights - 1);
	return lightNum;
}

__device__ __host__ glm::vec3 getPointOnSquarePlane(thrust::default_random_engine& rng, Geom light)
{
	thrust::uniform_real_distribution<float> u01(0, 1);
	float rand = u01(rng);
	glm::vec2 randpoint(u01(rng), u01(rng));
	glm::vec3 point_on_plane = glm::vec3((randpoint - glm::vec2(0.5f)), 0.f);
	//Transform this light to geom's local space 
	glm::vec3 point_on_plane_local = glm::vec3(light.transform * glm::vec4(point_on_plane, 1.f));
	return point_on_plane_local;
}

__global__ void shadeMaterialDirectLighting(
	int iter
	, int num_paths
	, ShadeableIntersection* shadeableIntersections
	, PathSegment* pathSegments
	, Material* materials
	, Geom* lights
	, int numLights
)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		ShadeableIntersection& currisect = shadeableIntersections[idx];
		PathSegment& currpath = pathSegments[idx];
		thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
		//thrust::uniform_real_distribution<float> u01(0, 1);

		if (currpath.remainingBounces !=  1 && currpath.remainingBounces > 0 && currisect.t > 0.f)
		{
			Material material = materials[currisect.materialId];
			glm::vec3 materialColor = material.color;

			// If the material indicates that the object was a light, "light" the ray
			// If object is light, stop bouncing
			if (material.emittance > 0.0f) {
				currpath.remainingBounces = 0;
				currpath.color *= (materialColor * material.emittance);
			}
			// Otherwise, do some pseudo-lighting computation. This is actually more
			// like what you would expect from shading in a rasterizer like OpenGL.
			else {
				if (currpath.remainingBounces != 0)
				{
					scatterRay(currpath,
						getPointOnRay(currpath.ray, currisect.t),
						currisect.surfaceNormal,
						material,
						rng);
				}
				else
				{
					currpath.color = glm::vec3(0.f);
				}

			}
			--currpath.remainingBounces;
			// If there was no intersection, color the ray black.
			// Lots of renderers use 4 channel color, RGBA, where A = alpha, often
			// used for opacity, in which case they can indicate "no opacity".
			// This can be useful for post-processing and image compositing.
		}
		else if (currpath.remainingBounces == 1 && currisect.t > 0.f)
		{
			Material material = materials[currisect.materialId];
			glm::vec3 materialColor = material.color;
			// If the material indicates that the object was a light, "light" the ray
			// If object is light, stop bouncing
			if (material.emittance > 0.0f) {
				currpath.remainingBounces = 0;
				currpath.color *= (materialColor * material.emittance);
			}
			// Otherwise, do some pseudo-lighting computation. This is actually more
			// like what you would expect from shading in a rasterizer like OpenGL.
			else
			{
				int randLight = getRandLightIdx(numLights, rng);
				glm::vec3 point_on_light = getPointOnSquarePlane(rng, lights[randLight]);
				//To set color 
				scatterRay(currpath
					, getPointOnRay(currpath.ray, currisect.t)
					, currisect.surfaceNormal
					, material
					, rng);
				//Ray should hit to the randomly selected light 
				if (!material.hasRefractive)
				{
					currpath.ray.direction = glm::normalize(point_on_light - currpath.ray.origin);
				}
				--currpath.remainingBounces;
			}		
		}
		else
		{
			currpath.remainingBounces = 0;
			currpath.color = glm::vec3(0.0f);
		}
	}
}


// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

//Helper for sorting rays by material 
struct checkMaterialID {
	__host__ __device__ bool operator()(const ShadeableIntersection& isect1, const ShadeableIntersection& isect2) {
		if (isect1.materialId < isect2.materialId) {
			return 1;
		}
		else return 0;
	}
};

//Helper for stream compacting rays 
struct pathTerminated {
	__host__ __device__ bool operator()(const PathSegment path) {
		if (path.remainingBounces > 0) {
			return 1;
		}
		else return 0;
	}
};

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter) {
	const int traceDepth = hst_scene->state.traceDepth;
	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

	///////////////////////////////////////////////////////////////////////////

	// Recap:
	// * Initialize array of path rays (using rays that come out of the camera)
	//   * You can pass the Camera object to that kernel.
	//   * Each path ray must carry at minimum a (ray, color) pair,
	//   * where color starts as the multiplicative identity, white = (1, 1, 1).
	//   * This has already been done for you.
	// * For each depth:
	//   * Compute an intersection in the scene for each path ray.
	//     A very naive version of this has been implemented for you, but feel
	//     free to add more primitives and/or a better algorithm.
	//     Currently, intersection distance is recorded as a parametric distance,
	//     t, or a "distance along the ray." t = -1.0 indicates no intersection.
	//     * Color is attenuated (multiplied) by reflections off of any object
	//   * TODO: Stream compact away all of the terminated paths.
	//     You may use either your implementation or `thrust::remove_if` or its
	//     cousins.
	//     * Note that you can't really use a 2D kernel launch any more - switch
	//       to 1D.
	//   * TODO: Shade the rays that intersected something or didn't bottom out.
	//     That is, color the ray by performing a color computation according
	//     to the shader, then generate a new ray to continue the ray path.
	//     We recommend just updating the ray's PathSegment in place.
	//     Note that this step may come before or after stream compaction,
	//     since some shaders you write may also cause a path to terminate.
	// * Finally, add this iteration's results to the image. This has been done
	//   for you.

	// TODO: perform one iteration of path tracing


	generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");

	//std::cout << "PATHTRACE : Num triangles in this mesh are" << scene->mesh.num_triangles << std::endl; 

	int depth = 0;
	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

	bool iterationComplete = false;
	while (!iterationComplete)
	{
		// clean shading chunks
		hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

		// tracing
		dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;

#ifdef CACHE_FIRST_BOUNCE
		if (depth == 0 && iter == 1)
		{
			//For the first iteration, compute the intersections in the cache buffer and copy them into the intersections buffer 
			computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
				depth
				, num_paths
				, dev_paths
				, dev_geoms
				, hst_scene->geoms.size()
				, dev_cache_intersections
				, dev_mesh_triangles
				, hst_scene->mesh.num_triangles
				, iter);
			checkCUDAError("First bounce cache error");
			hipDeviceSynchronize();
			hipMemcpy(dev_intersections, dev_cache_intersections, pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
			depth++;
		}
		if (depth == 0 && iter != 1)
		{
			//For all the other iterations, use the cached intersections instead of computing them again
			hipMemcpy(dev_intersections, dev_cache_intersections, pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
		}

		if (depth != 0)
		{
			computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
				depth
				, num_paths
				, dev_paths
				, dev_geoms
				, hst_scene->geoms.size()
				, dev_intersections
				, dev_mesh_triangles
				, hst_scene->mesh.num_triangles
				, iter);
			checkCUDAError("Error in trace one bounce");
			hipDeviceSynchronize();
			depth++;
	}
#else
		computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
			depth
			, num_paths
			, dev_paths
			, dev_geoms
			, hst_scene->geoms.size()
			, dev_intersections
			, dev_mesh_triangles
			, hst_scene->mesh.num_triangles
			, iter);
		checkCUDAError("trace one bounce");
		hipDeviceSynchronize();
		depth++;

#endif // CACHE_FIRST_BOUNCE

		//Ray sorting by material 
#ifdef SORT_RAYS_BY_MATERIALS 
		thrust::device_ptr<ShadeableIntersection> thrust_dev_isects(dev_intersections);
		thrust::device_ptr<PathSegment> thrust_dev_pathsegs(dev_paths);
		thrust::sort_by_key(thrust_dev_isects, thrust_dev_isects + num_paths, thrust_dev_pathsegs, checkMaterialID());
#endif // SORT_RAYS_BY_MATERIALS 


		// TODO:
		// --- Shading Stage ---
		// Shade path segments based on intersections and generate new rays by
		// evaluating the BSDF.
		// Start off with just a big kernel that handles all the different
		// materials you have in the scenefile.
		// TODO: compare between directly shading the path segments and shading
		// path segments that have been reshuffled to be contiguous in memory.
//#ifdef USE_SHADE_MATERIAL
//		shadeMaterial << <numblocksPathSegmentTracing, blockSize1d >> > (
//			iter,
//			num_paths,
//			dev_intersections,
//			dev_paths,
//			dev_materials
//			);
//#else
//		shadeFakeMaterial << <numblocksPathSegmentTracing, blockSize1d >> > (
//			iter,
//			num_paths,
//			dev_intersections,
//			dev_paths,
//			dev_materials
//			);
//		iterationComplete = true;
//#endif // USE_SHADE_MATERIAL

#ifdef DIRECT_LIGHTING
		shadeMaterialDirectLighting << <numblocksPathSegmentTracing, blockSize1d >> > (
			iter,
			num_paths,
			dev_intersections,
			dev_paths,
			dev_materials,
			dev_lights,
			hst_scene->lights.size()
			);
#else
		shadeMaterial << <numblocksPathSegmentTracing, blockSize1d >> > (
			iter,
			num_paths,
			dev_intersections,
			dev_paths,
			dev_materials
			);
#endif // DIRECT_LIGHTING



#ifdef STREAM_COMPACT_RAYS
		//Update the path ending 
		PathSegment* updated_path_end = thrust::stable_partition(thrust::device, dev_paths, dev_paths + num_paths, pathTerminated());
		num_paths = updated_path_end - dev_paths;

		//Set iteration complete based off stream compaction results.
		//Check if iteration is completed 
		if (num_paths <= 0)
		{
			iterationComplete = true;
			//printf("Iteration complete! \n"); 
		}
		depth++;
#endif // STREAM_COMPACT_RAYS
}

#ifdef USE_SHADE_MATERIAL
	num_paths = dev_path_end - dev_paths;
#endif // USE_SHADE_MATERIAL

	// Assemble this iteration and apply it to the image
	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather << <numBlocksPixels, blockSize1d >> > (num_paths, dev_image, dev_paths);

	///////////////////////////////////////////////////////////////////////////

	// Send results to OpenGL buffer for rendering
	sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_image);

	// Retrieve image from GPU
	hipMemcpy(hst_scene->state.image.data(), dev_image,
		pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	checkCUDAError("pathtrace");
	}