#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/partition.h>
#include <thrust/device_ptr.h>
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"
#include <>
#include <thrust/sort.h>
#include <chrono>
#include <iostream>
#include <iomanip>

#define ERRORCHECK 1
#define matSort false
#define CACHE_FIRST_BOUNCE false
#define stochasticAlias true
#define DIRECT_LIGHTING true
#define DOF false


#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
    getchar();
#  endif
    exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
    int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene* hst_scene = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...

static int* dev_materialIds;
static PathSegment* dev_cachedPaths = NULL;
static ShadeableIntersection* dev_cachedIntersections = NULL;

//static std::chrono::steady_clock::time_point timer;
//static ShadeableIntersection* dev_firstHit = NULL;

void pathtraceInit(Scene* scene) {
    hst_scene = scene;
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

    hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

    hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    // TODO: initialize any extra device memeory you need

    hipMalloc(&dev_materialIds, pixelcount * sizeof(int));
    hipMalloc(&dev_cachedPaths, pixelcount * sizeof(PathSegment));
    hipMalloc(&dev_cachedIntersections, pixelcount * sizeof(ShadeableIntersection));

}

void pathtraceFree() {
    hipFree(dev_image);  // no-op if dev_image is null
    hipFree(dev_paths);
    hipFree(dev_geoms);
    hipFree(dev_materials);
    hipFree(dev_intersections);
    // TODO: clean up any extra device memory you created
    hipFree(dev_materialIds);
    hipFree(dev_cachedPaths);
    hipFree(dev_cachedIntersections);
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);
        PathSegment& segment = pathSegments[index];

        segment.ray.origin = cam.position;
        segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

        // TODO: implement antialiasing by jittering the ray
        segment.ray.direction = glm::normalize(cam.view
            - cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
            - cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
        );

        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;
    }
}

__global__ void rayGenerator_withAntiAliasing(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    thrust::default_random_engine rng = makeSeededRandomEngine(x, y, 0);
    thrust::uniform_real_distribution<float> u(-1, 1);

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);
        PathSegment& segment = pathSegments[index];

        segment.ray.origin = cam.position;
        segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

        segment.ray.direction = glm::normalize(cam.view
            - cam.right * cam.pixelLength.x * ((float)x + u(rng) - (float)cam.resolution.x * 0.5f)
            - cam.up * cam.pixelLength.y * ((float)y + u(rng) - (float)cam.resolution.y * 0.5f)
        );

        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;


        
        if (DOF) {

            glm::vec3 focalPoint = segment.ray.origin + (8.0f * segment.ray.direction);
            glm::vec3 rand{ u(rng) / 2 , u(rng) / 2 , 0 };
            rand = rand * 0.5f;
            segment.ray.origin = segment.ray.origin + rand;
            segment.ray.direction = glm::normalize(focalPoint - segment.ray.origin);
        }

    }
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
    int depth
    , int num_paths
    , PathSegment* pathSegments
    , Geom* geoms
    , int geoms_size
    , ShadeableIntersection* intersections
)
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index < num_paths)
    {
        PathSegment pathSegment = pathSegments[path_index];

        float t;
        glm::vec3 intersect_point;
        glm::vec3 normal;
        float t_min = FLT_MAX;
        int hit_geom_index = -1;
        bool outside = true;

        glm::vec3 tmp_intersect;
        glm::vec3 tmp_normal;

        // naive parse through global geoms

        for (int i = 0; i < geoms_size; i++)
        {
            Geom& geom = geoms[i];

            if (geom.type == CUBE)
            {
                t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            else if (geom.type == SPHERE)
            {
                t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            else if (geom.type == TRIANGLE) {
                t = triangleIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            // TODO: add more intersection tests here... triangle? metaball? CSG?

            // Compute the minimum t from the intersection tests to determine what
            // scene geometry object was hit first.
            if (t > 0.0f && t_min > t)
            {
                t_min = t;
                hit_geom_index = i;
                intersect_point = tmp_intersect;
                normal = tmp_normal;
            }
        }

        if (hit_geom_index == -1)
        {
            intersections[path_index].t = -1.0f;
        }
        else
        {
            //The ray hits something
            intersections[path_index].t = t_min;
            intersections[path_index].materialId = geoms[hit_geom_index].materialid;
            intersections[path_index].surfaceNormal = normal;
        }
    }
}

__global__ void BSDFShader(int iter
    , int num_paths
    , ShadeableIntersection* shadeableIntersections
    , PathSegment* pathSegments
    , Material* materials
    , glm::vec3 globalLight)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_paths) { return; }

    ShadeableIntersection intersection = shadeableIntersections[idx];
    PathSegment segment = pathSegments[idx];

    // If there was no intersection, color the ray black.
    if (intersection.t <= 0.0f) {
        segment.remainingBounces = 0;
        segment.color = glm::vec3(0.0f, 0.0f, 0.0f);
        pathSegments[idx] = segment;
        return;
    }

    Material mat = materials[intersection.materialId];
    glm::vec3 matColor = mat.color;

    // If the material indicates that the object was a light, "light" the ray
    if (mat.emittance > 0.0f) {
        segment.color = segment.color * (matColor * mat.emittance);
        segment.remainingBounces = 0;
        pathSegments[idx] = segment;
        return;
    }

    // Else we handle the case that we hit a regular object
    // First we update the color of the path segment, then we can compute the new ray direction
    thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
    scatterRay(segment, getPointOnRay(segment.ray, intersection.t), intersection.surfaceNormal, mat, rng, iter, segment.remainingBounces);

    if (segment.remainingBounces == 0) {
        segment.color = segment.color * globalLight;
    }
    pathSegments[idx] = segment;
}


// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];
        image[iterationPath.pixelIndex] += iterationPath.color;
    }
}

struct isPathAlive {
    __host__ __device__
        bool operator()(const PathSegment& path) {
        return path.remainingBounces > 0;
    }
};

struct compareMat {
    __host__ __device__
        bool operator()(const ShadeableIntersection& MatA, const ShadeableIntersection& MatB) {
        return MatA.materialId > MatB.materialId;
    }
};

__global__ void getMatID(int paths, ShadeableIntersection* shadeableIntersections, int* result) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= paths) { return; }
    result[idx] = shadeableIntersections[idx].materialId;
}


/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter) {

    const int traceDepth = hst_scene->state.traceDepth;
    const Camera& cam = hst_scene->state.camera;
    glm::vec3 global_Light = hst_scene->globalLight;

    const int numPx = cam.resolution.x * cam.resolution.y;

    if (!DIRECT_LIGHTING) {
        global_Light = glm::vec3(0, 0, 0);
    }

    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    const int blockSize1d = 128;

        if (iter == 1 || !CACHE_FIRST_BOUNCE) {
        if (stochasticAlias) {
            rayGenerator_withAntiAliasing << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth, dev_paths);
        }
        else {
            generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth, dev_paths);
        }
    }
    else {
        hipMemcpy(dev_paths, dev_cachedPaths, numPx * sizeof(PathSegment), hipMemcpyDeviceToDevice);
        hipMemcpy(dev_intersections, dev_cachedIntersections, numPx * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
    }

    int depth = 0;
    PathSegment* dev_path_end = dev_paths + numPx;
    int num_paths = dev_path_end - dev_paths;
    int total_paths = num_paths;
    isBouncing data_compactor;

    bool iterating = false;
    while (!iterating) {

        dim3 blockCount = (num_paths + blockSize1d - 1) / blockSize1d;
        if (iter == 1 || !CACHE_FIRST_BOUNCE || depth > 0) {
            hipMemset(dev_intersections, 0, numPx * sizeof(ShadeableIntersection));

            computeIntersections << <blockCount, blockSize1d >> > (
                depth, num_paths, dev_paths, dev_geoms, hst_scene->geoms.size(), dev_intersections);
            hipDeviceSynchronize();

            if (matSort) {
                getMatID << <blockCount, blockSize1d >> > (num_paths, dev_intersections, dev_materialIds);
                thrust::device_ptr<ShadeableIntersection> dev_intersections_start(dev_intersections);
                thrust::device_ptr<PathSegment> dev_paths_start(dev_paths);
                thrust::device_ptr<int> dev_materialIds_start(dev_materialIds);
                thrust::zip_iterator<thrust::tuple<thrust::device_ptr<ShadeableIntersection>, thrust::device_ptr<PathSegment>>> zipped = thrust::make_zip_iterator(thrust::make_tuple(dev_intersections_start, dev_paths_start));
                thrust::sort_by_key(dev_materialIds_start, dev_materialIds_start + num_paths, zipped);
            }
        }

        if (iter == 1 && CACHE_FIRST_BOUNCE && depth == 0) {
            hipMemcpy(dev_cachedPaths, dev_paths, numPx * sizeof(PathSegment), hipMemcpyDeviceToDevice);
            hipMemcpy(dev_cachedIntersections, dev_intersections, numPx * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
        }

        BSDFShader << <blockCount, blockSize1d >> > (iter, num_paths, dev_intersections, dev_paths, dev_materials, global_Light);
        hipDeviceSynchronize();

        thrust::device_ptr<PathSegment> start(dev_paths);
        thrust::device_ptr<PathSegment> end(dev_path_end);
        end = thrust::partition(start, end, data_compactor);

        dev_path_end = thrust::raw_pointer_cast(end);
        num_paths = dev_path_end - dev_paths;
        depth++;

        if (num_paths == 0 || depth > traceDepth) {
            iterating = true;
        }
    }

    // Assemble this iteration and apply it to the image
    dim3 pxPerBlock = (numPx + blockSize1d - 1) / blockSize1d;
    finalGather << <pxPerBlock, blockSize1d >> > (total_paths, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
        numPx * sizeof(glm::vec3), hipMemcpyDeviceToHost);
}
