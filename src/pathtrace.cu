#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/partition.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"


constexpr bool
	sortByMaterial = false,
	cacheFirstBounce = false;


#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (hipSuccess == err) {
		return;
	}

	fprintf(stderr, "CUDA error");
	if (file) {
		fprintf(stderr, " (%s:%d)", file, line);
	}
	fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
	getchar();
#  endif
	exit(EXIT_FAILURE);
#endif
}

__host__ __device__ thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
	int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
		int iter, glm::vec3* image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.x && y < resolution.y) {
		int index = x + (y * resolution.x);
		glm::vec3 pix = image[index];

		glm::ivec3 color;
		color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
		color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
		color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = color.x;
		pbo[index].y = color.y;
		pbo[index].z = color.z;
	}
}

static Scene *hst_scene = nullptr;
static glm::vec3 *dev_image = nullptr;
static Geom *dev_geoms = nullptr;
static Material *dev_materials = nullptr;
static PathSegment *dev_paths = nullptr;
static ShadeableIntersection *dev_intersections = nullptr;
static AABBTreeNode *dev_aabbTree = nullptr;
static int aabbTreeRoot;

// static variables for device memory, any extra info you need, etc

static bool firstBounceCached = false;
static ShadeableIntersection *dev_firstBounceIntersections = nullptr;

void pathtraceInit(Scene *scene) {
	hst_scene = scene;
	const Camera &cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	// initialize any extra device memeory you need
	if (cacheFirstBounce) {
		hipMalloc(&dev_firstBounceIntersections, pixelcount * sizeof(ShadeableIntersection));
	}
	firstBounceCached = false;

	hipMalloc(&dev_aabbTree, scene->aabbTree.size() * sizeof(AABBTreeNode));
	hipMemcpy(dev_aabbTree, scene->aabbTree.data(), scene->aabbTree.size() * sizeof(AABBTreeNode), hipMemcpyHostToDevice);

	aabbTreeRoot = scene->aabbTreeRoot;

	checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
	hipFree(dev_image);  // no-op if dev_image is null
	hipFree(dev_paths);
	hipFree(dev_geoms);
	hipFree(dev_materials);
	hipFree(dev_intersections);

	// clean up any extra device memory you created
	if (cacheFirstBounce) {
		hipFree(dev_firstBounceIntersections);
	}
	hipFree(dev_aabbTree);

	checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment *pathSegments) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x >= cam.resolution.x && y >= cam.resolution.y) {
		return;
	}


	int index = x + (y * cam.resolution.x);
	PathSegment &segment = pathSegments[index];

	thrust::default_random_engine rand = makeSeededRandomEngine(iter, index, -1);
	thrust::uniform_real_distribution<float> dist(-0.5f, 0.5f);

	segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

	// implement antialiasing by jittering the ray
	glm::vec3 dir =
		cam.view -
		cam.right * (cam.pixelLength.x * ((static_cast<float>(x) + dist(rand)) / cam.resolution.x - 0.5f)) -
		cam.up * (cam.pixelLength.y * ((static_cast<float>(y) + dist(rand)) / cam.resolution.x - 0.5f));

	// depth of field
	dir *= cam.focalDistance;
	glm::vec2 aperture = sampleUnitDiskUniform(rand) * cam.aperture;
	glm::vec3 dofOffset = aperture.x * cam.right + aperture.y * cam.up;

	segment.ray.origin = cam.position + dofOffset;
	segment.ray.direction = glm::normalize(dir - dofOffset);

	segment.pixelIndex = index;
	segment.remainingBounces = traceDepth;
}

__device__ float max3(glm::vec3 xyz) {
	return glm::max(xyz.x, glm::max(xyz.y, xyz.z));
}
__device__ float min3(glm::vec3 xyz) {
	return glm::min(xyz.x, glm::min(xyz.y, xyz.z));
}
__device__ bool rayBoxIntersection(const Ray &ray, glm::vec3 min, glm::vec3 max, float far) {
	min = (min - ray.origin) / ray.direction;
	max = (max - ray.origin) / ray.direction;
	float rmin = max3(glm::min(min, max)), rmax = min3(glm::max(min, max));
	return rmin < far && rmax >= rmin && rmax > 0.0f;
}

__device__ bool rayGeomIntersection(const Ray &ray, const Geom &geom, float *dist, glm::vec3 *normal) {
	float t = -1.0f;
	glm::vec3 norm;
	if (geom.type == GeomType::CUBE) {
		t = boxIntersectionTest(geom.implicit, ray, norm);
	} else if (geom.type == GeomType::SPHERE) {
		t = sphereIntersectionTest(geom.implicit, ray, norm);
	} else if (geom.type == GeomType::TRIANGLE) {
		glm::vec2 bary;
		t = triangleIntersectionTest(geom.triangle, ray, &bary);
		if (t >= 0.0f) {
			norm =
				geom.triangle.normals[0] * (1.0f - bary.x - bary.y) +
				geom.triangle.normals[1] * bary.x + geom.triangle.normals[2] * bary.y;
			norm = glm::normalize(norm);
		}
	}
	if (t > 0.0f && t < *dist) {
		*dist = t;
		*normal = norm;
		return true;
	}
	return false;
}

__device__ int traverseAABBTree(
	const Ray &ray, const AABBTreeNode *tree, int root, const Geom *geoms, float *dist, glm::vec3 *normal
) {
	constexpr int geomTestInterval = 4;

	int stack[64], top = 1;
	stack[0] = root;
	int candidates[geomTestInterval * 2], numCandidates = 0;
	int counter = 0, resIndex = -1;
	while (top > 0) {
		const AABBTreeNode &node = tree[stack[--top]];
		bool
			leftIsect = rayBoxIntersection(ray, node.leftAABBMin, node.leftAABBMax, *dist),
			rightIsect = rayBoxIntersection(ray, node.rightAABBMin, node.rightAABBMax, *dist);
		if (leftIsect) {
			if (node.leftChild < 0) {
				candidates[numCandidates++] = ~node.leftChild;
			} else {
				stack[top++] = node.leftChild;
			}
		}
		if (rightIsect) {
			if (node.rightChild < 0) {
				candidates[numCandidates++] = ~node.rightChild;
			} else {
				stack[top++] = node.rightChild;
			}
		}

		if (++counter == geomTestInterval) {
			for (int i = 0; i < numCandidates; ++i) {
				if (rayGeomIntersection(ray, geoms[candidates[i]], dist, normal)) {
					resIndex = candidates[i];
				}
			}
			numCandidates = 0;
			counter = 0;
		}
	}
	for (int i = 0; i < numCandidates; ++i) {
		if (rayGeomIntersection(ray, geoms[candidates[i]], dist, normal)) {
			resIndex = candidates[i];
		}
	}
	return resIndex;
}

// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
	int depth, int num_paths, PathSegment *pathSegments,
	const Geom *geoms, int geoms_size, const AABBTreeNode *aabbTree, int aabbTreeRoot,
	ShadeableIntersection *intersections
) {
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index >= num_paths) {
		return;
	}

	PathSegment pathSegment = pathSegments[path_index];

	glm::vec3 normal;
	float t_min = FLT_MAX;
	int hit_geom_index = -1;

	hit_geom_index = traverseAABBTree(pathSegment.ray, aabbTree, aabbTreeRoot, geoms, &t_min, &normal);

	if (hit_geom_index == -1)
	{
		intersections[path_index].t = -1.0f;
		intersections[path_index].materialId = -1;
	}
	else
	{
		//The ray hits something
		intersections[path_index].t = t_min;
		intersections[path_index].materialId = geoms[hit_geom_index].materialid;
		intersections[path_index].surfaceNormal = normal;
	}
}

__global__ void shade(
	int iter, int depth, int num_paths,
	ShadeableIntersection *intersections, PathSegment *paths, Material *materials
) {
	int iSelf = blockIdx.x * blockDim.x + threadIdx.x;
	if (iSelf >= num_paths) {
		return;
	}

	ShadeableIntersection intersection = intersections[iSelf];
	PathSegment path = paths[iSelf];

	if (intersection.materialId != -1) {
		/*path.color = (intersection.surfaceNormal + 1.0f) * 0.5f;
		path.remainingBounces = -1;*/

		thrust::default_random_engine rng = makeSeededRandomEngine(iter, iSelf, depth);
		scatterRay(
			path, path.ray.origin + path.ray.direction * intersection.t, intersection.surfaceNormal,
			materials[intersection.materialId], rng
		);
	} else {
		path.color = glm::vec3(0.0f);
		path.remainingBounces = 0;
	}
	paths[iSelf] = path;
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 * image, PathSegment * iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths) {
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

struct IsRayTravelling {
	__host__ __device__ bool operator()(const PathSegment &path) {
		return path.remainingBounces > 0;
	}
};

struct NoContribution {
	__host__ __device__ bool operator()(const PathSegment &path) {
		return path.remainingBounces != -1;
	}
};

struct MaterialCompare {
	__host__ __device__ bool operator()(const ShadeableIntersection &lhs, const ShadeableIntersection &rhs) {
		return lhs.materialId > rhs.materialId;
	}
};

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4 *pbo, int frame, int iter) {
	const int traceDepth = hst_scene->state.traceDepth;
	const Camera &cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
			(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
			(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

	///////////////////////////////////////////////////////////////////////////

	// Recap:
	// * Initialize array of path rays (using rays that come out of the camera)
	//   * You can pass the Camera object to that kernel.
	//   * Each path ray must carry at minimum a (ray, color) pair,
	//   * where color starts as the multiplicative identity, white = (1, 1, 1).
	//   * This has already been done for you.
	// * For each depth:
	//   * Compute an intersection in the scene for each path ray.
	//     A very naive version of this has been implemented for you, but feel
	//     free to add more primitives and/or a better algorithm.
	//     Currently, intersection distance is recorded as a parametric distance,
	//     t, or a "distance along the ray." t = -1.0 indicates no intersection.
	//     * Color is attenuated (multiplied) by reflections off of any object
	//   * Stream compact away all of the terminated paths.
	//     You may use either your implementation or `thrust::remove_if` or its
	//     cousins.
	//     * Note that you can't really use a 2D kernel launch any more - switch
	//       to 1D.
	//   * Shade the rays that intersected something or didn't bottom out.
	//     That is, color the ray by performing a color computation according
	//     to the shader, then generate a new ray to continue the ray path.
	//     We recommend just updating the ray's PathSegment in place.
	//     Note that this step may come before or after stream compaction,
	//     since some shaders you write may also cause a path to terminate.
	// * Finally, add this iteration's results to the image. This has been done
	//   for you.

	generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");

	PathSegment* dev_path_end = dev_paths + pixelcount;
	int num_paths = dev_path_end - dev_paths;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks

	for (int depth = 0; num_paths > 0; ++depth) {
		// clean shading chunks
		hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

		// tracing
		int numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
		if (depth == 0 && firstBounceCached) {
			hipMemcpy(
				dev_intersections, dev_firstBounceIntersections,
				pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice
			);
		} else {
			computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>>(
				depth, num_paths, dev_paths,
				dev_geoms, hst_scene->geoms.size(), dev_aabbTree, aabbTreeRoot,
				dev_intersections
			);
			checkCUDAError("trace one bounce");

			if (cacheFirstBounce && depth == 0) {
				hipMemcpy(
					dev_firstBounceIntersections, dev_intersections,
					pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice
				);
				firstBounceCached = true;
			}
		}

		// --- Shading Stage ---
		// Shade path segments based on intersections and generate new rays by
		// evaluating the BSDF.
		// Start off with just a big kernel that handles all the different
		// materials you have in the scenefile.
		// compare between directly shading the path segments and shading
		// path segments that have been reshuffled to be contiguous in memory.

		if (sortByMaterial) {
			thrust::sort_by_key(
				thrust::device, dev_intersections, dev_intersections + num_paths, dev_paths, MaterialCompare()
			);
		}

		shade<<<numblocksPathSegmentTracing, blockSize1d>>>(
			iter, depth, num_paths, dev_intersections, dev_paths, dev_materials
		);

		num_paths = thrust::partition(thrust::device, dev_paths, dev_paths + num_paths, IsRayTravelling()) - dev_paths;
	}

	// Assemble this iteration and apply it to the image
	int gather_paths = thrust::remove_if(thrust::device, dev_paths, dev_path_end, NoContribution()) - dev_paths;
	if (gather_paths > 0) {
		dim3 numBlocksPixels = (gather_paths + blockSize1d - 1) / blockSize1d;
		finalGather<<<numBlocksPixels, blockSize1d>>>(gather_paths, dev_image, dev_paths);
	}

	///////////////////////////////////////////////////////////////////////////

	// Send results to OpenGL buffer for rendering
	sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

	// Retrieve image from GPU
	hipMemcpy(hst_scene->state.image.data(), dev_image,
			pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	checkCUDAError("pathtrace");
}
