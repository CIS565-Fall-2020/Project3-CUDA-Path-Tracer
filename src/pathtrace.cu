#include "hip/hip_runtime.h"
﻿#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <iostream>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/partition.h>
#include <thrust/sort.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"
#include "warpfunctions.h"


#define BOUNDINGBOXINTERSECTIONTEST true
#define DEPTHOFFIELD false
#define ANTIALIASING true
#define CACHEFIRSTBOUNCE !ANTIALIASING
#define DIRECTLIGHTING true
#define MOTIONBLUR false

#define ERRORCHECK 1
#define RECORDEDITERATION 100
#define MOTIONBLUR_VELOCITY glm::vec3(0, 0.96f, 0)

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)


void checkCUDAErrorFn(const char* msg, const char* file, int line)
{
#if ERRORCHECK
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (hipSuccess == err) {
		return;
	}

	fprintf(stderr, "CUDA error");
	if (file) {
		fprintf(stderr, " (%s:%d)", file, line);
	}
	fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
	getchar();
#  endif
	exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) 
{
	int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
							   int iter, glm::vec3* image)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.x && y < resolution.y)
	{
		int index = x + (y * resolution.x);
		glm::vec3 pix = image[index];

		glm::ivec3 color;
		color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
		color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
		color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = color.x;
		pbo[index].y = color.y;
		pbo[index].z = color.z;
	}
}

static float gpu_time_accumulator = 0.0f;

static Scene* hst_scene = nullptr;
static glm::vec3* dev_image = nullptr;
static Geom* dev_geoms = nullptr;
static Geom* dev_light_geoms = nullptr;
static Material* dev_materials = nullptr;
static PathSegment* dev_paths = nullptr;
static ShadeableIntersection* dev_intersections = nullptr;

// Extra static variables for device memory, declared here by me 
static PathSegment* dev_first_paths = nullptr;
static ShadeableIntersection* dev_first_intersections = nullptr;

// gltf mesh data
static float* dev_gltf_vertices = nullptr;                  
static unsigned int* dev_gltf_faces = nullptr;
static unsigned int* dev_gltf_verts_offset = nullptr;
static unsigned int* dev_gltf_faces_offset = nullptr;
static float* dev_gltf_bbox_verts = nullptr;


hipEvent_t iter_event_start = nullptr;
hipEvent_t iter_event_end = nullptr;

void pathtraceInit(Scene* scene) 
{
	hst_scene = scene;
	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// if glTF mesh exists
	if (!scene->meshes.empty())
	{
		preprocessGltfData(scene);
	}

	hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	// Initialize the extra device memeory 
	hipMalloc(&dev_first_paths, pixelcount * sizeof(PathSegment));

	hipMalloc(&dev_first_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_first_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	hipMalloc(&dev_light_geoms, scene->lightGeoms.size() * sizeof(Geom));
	hipMemcpy(dev_light_geoms, scene->lightGeoms.data(), scene->lightGeoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

	hipEventCreate(&iter_event_start);
	hipEventCreate(&iter_event_end);

	checkCUDAError("pathtraceInit");
}

void pathtraceFree() 
{
	hipFree(dev_image);  // no-op if dev_image is null
	hipFree(dev_paths);
	hipFree(dev_geoms);
	hipFree(dev_materials);
	hipFree(dev_intersections);
	
	// Clean up those extra device variables 
	hipFree(dev_first_paths);
	hipFree(dev_first_intersections);

	hipFree(dev_gltf_faces);
	hipFree(dev_gltf_vertices);
	hipFree(dev_gltf_faces_offset);
	hipFree(dev_gltf_verts_offset);
	hipFree(dev_gltf_bbox_verts);

	checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) 
	{
		int index = x + (y * cam.resolution.x);
		PathSegment& segment = pathSegments[index];
		segment.color = glm::vec3(1.0f, 1.0f, 1.0f);
		
		// Set up the RNG
		thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, traceDepth);
		thrust::uniform_real_distribution<float> u01(0, 1);

#if ANTIALIASING
		// Do antialiasing by jittering the ray
		segment.ray = cam.rayCast(x + u01(rng), y + u01(rng));
#else
		segment.ray = cam.rayCast(x, y);
#endif // ANTIALIASING
		
#if DEPTHOFFIELD
		if (cam.lensRadius > 0)
		{
			// Sample point on lens
			glm::vec2 pLens = cam.lensRadius * WarpFunctions::squareToDiskConcentric(glm::vec2(u01(rng), u01(rng)));
			// Compute point on plane of focus
			glm::vec3 pFocus = segment.ray.origin + cam.focalDist * segment.ray.direction;
			// Update ray for effect of lens
			segment.ray.origin += glm::vec3(pLens.x, pLens.y, 0);
			segment.ray.direction = glm::normalize(pFocus - segment.ray.origin); 
		}
#endif // DEPTHOFFIELD

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(int iter,
									 int depth, 
									 int num_paths, 
									 PathSegment* pathSegments, 
									 Geom* geoms, 
									 int geoms_size, 
									 ShadeableIntersection* intersections,
								     unsigned int* faces,
									 float* vertices,
									 unsigned int* num_faces,
									 unsigned int* num_vertices,
									 float* bbox_verts
									 )
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		const PathSegment& pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms
		for (int i = 0; i < geoms_size; i++)
		{
			Geom& geom = geoms[i];

			if (geom.type == GeomType::CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == GeomType::SPHERE)
			{
				Ray tempRay = pathSegment.ray;
#if MOTIONBLUR
				thrust::default_random_engine rng = makeSeededRandomEngine(iter, path_index, pathSegment.remainingBounces);
				thrust::uniform_real_distribution<float> u01(0, 1);
				tempRay.origin -= cos((2 * u01(rng) - 1) * PI) * MOTIONBLUR_VELOCITY;
#endif // MOTIONBLUR
				t = sphereIntersectionTest(geom, tempRay, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == GeomType::MESH)
			{
				bool bbox_hit = true;
#if BOUNDINGBOXINTERSECTIONTEST
				int i = geom.meshid;
				Geom bbox_geom;
				bbox_geom.type = GeomType::CUBE;
				glm::vec3 bbox_scale(bbox_verts[i * 6 + 3] - bbox_verts[i * 6 + 0],
									 bbox_verts[i * 6 + 4] - bbox_verts[i * 6 + 1],
									 bbox_verts[i * 6 + 5] - bbox_verts[i * 6 + 2]);

				setGeomTransform(&bbox_geom, geom.transform * getTansformation(glm::vec3(0), glm::vec3(0), bbox_scale));
				t = boxIntersectionTest(bbox_geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
				// Do bounding box intersection culling if not
				bbox_hit = t > 0.0f && t < t_min;
#endif // BOUNDINGBOXINTERSECTIONTEST
				if (bbox_hit)
				{
					t = meshIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside,
											 faces, vertices, num_faces, num_vertices, bbox_verts);
				}
			}
			// TODO: add more intersection tests here... triangle? metaball? CSG?

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t < t_min)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			// The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].point = getPointOnRay(pathSegment.ray, t_min);
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;
			intersections[path_index].hitGeom = &geoms[hit_geom_index];
		}
	}
}

void preprocessGltfData(Scene* scene)
{
	int num_meshes = scene->getMeshesSize();

	hipMalloc(&dev_gltf_faces, scene->total_faces * sizeof(unsigned int));
	hipMalloc(&dev_gltf_vertices, scene->total_vertices * sizeof(float));
	hipMalloc(&dev_gltf_bbox_verts, 6 * num_meshes * sizeof(float));

	for (int i = 0, face_offset = 0, vertice_offset = 0; i < num_meshes; i++)
	{
		const gltf::Mesh<float>& mesh = scene->meshes[i];
		int cur_num_faces = mesh.faces.size();
		int cur_num_vertices = mesh.vertices.size();

		hipMemcpy(dev_gltf_faces + face_offset, mesh.faces.data(), cur_num_faces * sizeof(unsigned int),
			hipMemcpyKind::hipMemcpyHostToDevice);
		hipMemcpy(dev_gltf_vertices + vertice_offset, mesh.vertices.data(), cur_num_vertices * sizeof(float),
			hipMemcpyKind::hipMemcpyHostToDevice);
		hipMemcpy(dev_gltf_bbox_verts + i * 6, mesh.bbox_verts.data(), 6 * sizeof(float),
			hipMemcpyKind::hipMemcpyHostToDevice);
		
		scene->faces_per_mesh.push_back(face_offset);
		scene->vertices_per_mesh.push_back(vertice_offset);

		face_offset += cur_num_faces;
		vertice_offset += cur_num_vertices;
	}

	hipMalloc(&dev_gltf_verts_offset, scene->vertices_per_mesh.size() * sizeof(unsigned int));
	hipMalloc(&dev_gltf_faces_offset, scene->faces_per_mesh.size() * sizeof(unsigned int));

	hipMemcpy(dev_gltf_verts_offset, scene->vertices_per_mesh.data(), scene->vertices_per_mesh.size() * sizeof(unsigned int),
		hipMemcpyKind::hipMemcpyHostToDevice);
	hipMemcpy(dev_gltf_faces_offset, scene->faces_per_mesh.data(), scene->faces_per_mesh.size() * sizeof(unsigned int),
		hipMemcpyKind::hipMemcpyHostToDevice);

	checkCUDAError("preprocess gltf data");
}

__global__ void shadeMaterial(int iter,
							  int num_paths,
							  ShadeableIntersection* shadeableIntersections,
							  PathSegment* pathSegments,
							  Material* materials,
							  Geom* lightGeoms,
							  int num_lights)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) // if the intersection exists...
		{	// Set up the RNG
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
			thrust::uniform_real_distribution<float> u01(0, 1);

			Material material = materials[intersection.materialId];
			glm::vec3 materialColor = material.color;

			// If the material indicates that the object was a light, "light" the ray
			if (material.emittance > 0.0f)
			{
				pathSegments[idx].color *= (materialColor * material.emittance);
				pathSegments[idx].remainingBounces = 0;
			}
			else
			{
#if DIRECTLIGHTING
				scatterDirectRay(pathSegments[idx], intersection, material, rng, lightGeoms, num_lights);
#else
				scatterIndirectRay(pathSegments[idx], intersection, material, rng);
#endif // DIRECTLIGHTING
			}
		}
		else
		{// If there was no intersection, color the ray black.
			pathSegments[idx].color = glm::vec3(0.0f);
			pathSegments[idx].remainingBounces = 0;
		}
	}
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter) 
{
	const int traceDepth = hst_scene->state.traceDepth;
	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d((cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
							   (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

	float iter_time = 0.f;
	hipEventRecord(iter_event_start);
#if CACHEFIRSTBOUNCE
	if (iter == 1)
	{
		generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths);
		checkCUDAError("generate camera ray");
		hipMemcpy(dev_first_paths, dev_paths, 
				   pixelcount * sizeof(PathSegment), hipMemcpyKind::hipMemcpyDeviceToDevice);
		checkCUDAError("copy first paths to dev_first_paths");
	}
	else
	{
		hipMemcpy(dev_paths, dev_first_paths,
				   pixelcount * sizeof(PathSegment), hipMemcpyKind::hipMemcpyDeviceToDevice);
		checkCUDAError("get first paths from cache");
	}
#else
	generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");
#endif // CACHEFIRSTBOUNCE

	int depth = 0;
	PathSegment* dev_paths_end = dev_paths + pixelcount;
	int num_paths = dev_paths_end - dev_paths;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks
	for (int cur_num_paths = num_paths; cur_num_paths > 0; cur_num_paths = dev_paths_end - dev_paths)
	{
		// Clean shading chunks
		hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));
		dim3 numblocksPathSegmentTracing = (cur_num_paths + blockSize1d - 1) / blockSize1d;

		// Tracing
#if CACHEFIRSTBOUNCE
		if (depth == 0 && iter > 1)
		{
			hipMemcpy(dev_intersections, dev_first_intersections,
					   pixelcount * sizeof(ShadeableIntersection), hipMemcpyKind::hipMemcpyDeviceToDevice);
		}
		else
		{
			computeIntersections <<<numblocksPathSegmentTracing, blockSize1d >> > (
				iter,
				depth,
				cur_num_paths,
				dev_paths,
				dev_geoms,
				hst_scene->geoms.size(),
				dev_intersections,
				dev_gltf_faces,
				dev_gltf_vertices，
				dev_gltf_faces_offset,
				dev_gltf_verts_offset,
				dev_gltf_bbox_verts
			);

			// In the first bounce, store first intersections in the cache _dev_first_intersections_ 
			if (depth == 0 && iter == 1)
			{
				hipMemcpy(dev_first_intersections, dev_intersections,
						   pixelcount * sizeof(ShadeableIntersection), hipMemcpyKind::hipMemcpyDeviceToDevice);
			}
		}
#else
		computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
			iter,
			depth,
			cur_num_paths,
			dev_paths,
			dev_geoms,
			hst_scene->geoms.size(),
			dev_intersections,
			dev_gltf_faces,
			dev_gltf_vertices，
			dev_gltf_faces_offset,
			dev_gltf_verts_offset,
			dev_gltf_bbox_verts
		);
#endif // CACHEFIRSTBOUNCE

		checkCUDAError("trace one bounce");
		hipDeviceSynchronize();
		depth++;

		// --- Shading Stage ---
		// Before shading, sort the  pathSegments so that pathSegments with the same material are contiguous in memory 
		thrust::sort_by_key(thrust::device, dev_intersections, dev_intersections + cur_num_paths, dev_paths, material_comp());

		// Shade path segments based on intersections and generate new rays by evaluating the BSDF.
		shadeMaterial << <numblocksPathSegmentTracing, blockSize1d >> > (
			iter,
			cur_num_paths,
			dev_intersections,
			dev_paths,
			dev_materials,
			dev_light_geoms,
			hst_scene->lightGeoms.size()
		);

		// Stream compact away all of the terminated paths.
		dev_paths_end = thrust::partition(thrust::device, dev_paths, dev_paths_end, raytracing_continuing());
	}

	// Assemble this iteration and apply it to the image
	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather<<<numBlocksPixels, blockSize1d>>>(num_paths, dev_image, dev_paths);

	// Send results to OpenGL buffer for rendering
	sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

	// Calculate how long to finish this iteration
	hipEventRecord(iter_event_end);
	hipEventSynchronize(iter_event_end);
	hipEventElapsedTime(&iter_time, iter_event_start, iter_event_end);
	gpu_time_accumulator += iter_time;

	if (iter == RECORDEDITERATION)
	{
		std::cout << "Elapsed time to finish " << RECORDEDITERATION << " iterations: " << gpu_time_accumulator << "ms" << endl;
		std::cout << "Average time to run a single iteration: " << gpu_time_accumulator / RECORDEDITERATION << "ms" << endl;
	}

	// Retrieve image from GPU
	hipMemcpy(hst_scene->state.image.data(), dev_image, pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	checkCUDAError("pathtrace");
}