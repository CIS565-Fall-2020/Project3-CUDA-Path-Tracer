#include "hip/hip_runtime.h"
﻿#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/partition.h>
#include <thrust/sort.h>
#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1
#define CACHEFIRSTBOUNCE true
#define USEMATERIALSORT true

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line)
{
#if ERRORCHECK
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if (hipSuccess == err) {
		return;
	}

	fprintf(stderr, "CUDA error");
	if (file) {
		fprintf(stderr, " (%s:%d)", file, line);
	}
	fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
	getchar();
#  endif
	exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) 
{
	int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
							   int iter, glm::vec3* image)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < resolution.x && y < resolution.y)
	{
		int index = x + (y * resolution.x);
		glm::vec3 pix = image[index];

		glm::ivec3 color;
		color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
		color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
		color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = color.x;
		pbo[index].y = color.y;
		pbo[index].z = color.z;
	}
}

static Scene* hst_scene = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;

// Extra static variables for device memory, declared here by me 
static PathSegment* dev_first_paths = NULL;
static ShadeableIntersection* dev_first_intersections = NULL;

void pathtraceInit(Scene* scene) 
{
	hst_scene = scene;
	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
	hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

	hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

	hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
	hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

	hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
	hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

	hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	// Initialize the extra device memeory 
	hipMalloc(&dev_first_paths, pixelcount * sizeof(PathSegment));

	hipMalloc(&dev_first_intersections, pixelcount * sizeof(ShadeableIntersection));
	hipMemset(dev_first_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

	checkCUDAError("pathtraceInit");
}

void pathtraceFree() 
{
	hipFree(dev_image);  // no-op if dev_image is null
	hipFree(dev_paths);
	hipFree(dev_geoms);
	hipFree(dev_materials);
	hipFree(dev_intersections);
	
	// Clean up those extra device variables 
	hipFree(dev_first_paths);
	hipFree(dev_first_intersections);

	checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < cam.resolution.x && y < cam.resolution.y) 
	{
		int index = x + (y * cam.resolution.x);
		PathSegment& segment = pathSegments[index];

		segment.ray.origin = cam.position;
		segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

		// TODO: implement antialiasing by jittering the ray
		segment.ray.direction = glm::normalize(cam.view
			- cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
			- cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
		);

		segment.pixelIndex = index;
		segment.remainingBounces = traceDepth;
	}
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(int depth, 
									 int num_paths, 
									 PathSegment* pathSegments, 
									 Geom* geoms, 
									 int geoms_size, 
									 ShadeableIntersection* intersections)
{
	int path_index = blockIdx.x * blockDim.x + threadIdx.x;

	if (path_index < num_paths)
	{
		const PathSegment& pathSegment = pathSegments[path_index];

		float t;
		glm::vec3 intersect_point;
		glm::vec3 normal;
		float t_min = FLT_MAX;
		int hit_geom_index = -1;
		bool outside = true;

		glm::vec3 tmp_intersect;
		glm::vec3 tmp_normal;

		// naive parse through global geoms
		for (int i = 0; i < geoms_size; i++)
		{
			Geom& geom = geoms[i];

			if (geom.type == CUBE)
			{
				t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			else if (geom.type == SPHERE)
			{
				t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
			}
			// TODO: add more intersection tests here... triangle? metaball? CSG?

			// Compute the minimum t from the intersection tests to determine what
			// scene geometry object was hit first.
			if (t > 0.0f && t_min > t)
			{
				t_min = t;
				hit_geom_index = i;
				intersect_point = tmp_intersect;
				normal = tmp_normal;
			}
		}

		if (hit_geom_index == -1)
		{
			intersections[path_index].t = -1.0f;
		}
		else
		{
			//The ray hits something
			intersections[path_index].t = t_min;
			intersections[path_index].materialId = geoms[hit_geom_index].materialid;
			intersections[path_index].surfaceNormal = normal;
		}
	}
}

__global__ void shadeMaterial(int iter,
							  int num_paths,
							  ShadeableIntersection* shadeableIntersections,
							  PathSegment* pathSegments,
							  Material* materials)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < num_paths)
	{
		ShadeableIntersection intersection = shadeableIntersections[idx];
		if (intersection.t > 0.0f) // if the intersection exists...
		{	// Set up the RNG
			thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
			thrust::uniform_real_distribution<float> u01(0, 1);

			Material material = materials[intersection.materialId];
			glm::vec3 materialColor = material.color;

			// If the material indicates that the object was a light, "light" the ray
			if (material.emittance > 0.0f)
			{
				pathSegments[idx].color *= (materialColor * material.emittance);
				pathSegments[idx].remainingBounces = 0;
			}
			else
			{
				scatterRay(pathSegments[idx], getPointOnRay(pathSegments[idx].ray, intersection.t), intersection.surfaceNormal, material, rng);
			}
		}
		else
		{// If there was no intersection, color the ray black.
			pathSegments[idx].color = glm::vec3(0.0f);
			pathSegments[idx].remainingBounces = 0;
		}
	}
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < nPaths)
	{
		PathSegment iterationPath = iterationPaths[index];
		image[iterationPath.pixelIndex] += iterationPath.color;
	}
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter) 
{
	const int traceDepth = hst_scene->state.traceDepth;
	const Camera& cam = hst_scene->state.camera;
	const int pixelcount = cam.resolution.x * cam.resolution.y;

	// 2D block for generating ray from camera
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
		(cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

	// 1D block for path tracing
	const int blockSize1d = 128;

#if CACHEFIRSTBOUNCE
	if (iter == 1)
	{
		generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths);
		checkCUDAError("generate camera ray");
		hipMemcpy(dev_first_paths, dev_paths, 
				   pixelcount * sizeof(PathSegment), hipMemcpyKind::hipMemcpyDeviceToDevice);
		checkCUDAError("copy first paths to dev_first_paths");
	}
	else
	{
		hipMemcpy(dev_paths, dev_first_paths,
				   pixelcount * sizeof(PathSegment), hipMemcpyKind::hipMemcpyDeviceToDevice);
		checkCUDAError("get first paths from cache");
	}
#else
	generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth, dev_paths);
	checkCUDAError("generate camera ray");
#endif // CACHEFIRSTBOUNCE

	int depth = 0;
	PathSegment* dev_paths_end = dev_paths + pixelcount;
	int num_paths = dev_paths_end - dev_paths;

	// --- PathSegment Tracing Stage ---
	// Shoot ray into scene, bounce between objects, push shading chunks
	for (int cur_num_paths = num_paths; cur_num_paths > 0; cur_num_paths = dev_paths_end - dev_paths)
	{
		// Clean shading chunks
		hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));
		dim3 numblocksPathSegmentTracing = (cur_num_paths + blockSize1d - 1) / blockSize1d;

		// Tracing
#if CACHEFIRSTBOUNCE
		if (depth == 0 && iter > 1)
		{
			hipMemcpy(dev_intersections, dev_first_intersections,
					   pixelcount * sizeof(ShadeableIntersection), hipMemcpyKind::hipMemcpyDeviceToDevice);
		}
		else
		{
			computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
				depth,
				cur_num_paths,
				dev_paths,
				dev_geoms,
				hst_scene->geoms.size(),
				dev_intersections
			);

			// In the first bounce, store first intersections in the cache _dev_first_intersections_ 
			if (depth == 0 && iter == 1)
			{
				hipMemcpy(dev_first_intersections, dev_intersections,
						   pixelcount * sizeof(ShadeableIntersection), hipMemcpyKind::hipMemcpyDeviceToDevice);
			}
		}
#else
		computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
			depth,
			cur_num_paths,
			dev_paths,
			dev_geoms,
			hst_scene->geoms.size(),
			dev_intersections
		);
#endif // CACHEFIRSTBOUNCE

		checkCUDAError("trace one bounce");
		hipDeviceSynchronize();
		depth++;

		// --- Shading Stage ---
		// Before shading, sort the  pathSegments so that pathSegments with the same material are contiguous in memory 
#if USEMATERIALSORT
		thrust::sort_by_key(thrust::device, dev_intersections, dev_intersections + cur_num_paths, dev_paths, material_comp());
#endif // USEMATERIALSORT


		

		// Shade path segments based on intersections and generate new rays by evaluating the BSDF.
		shadeMaterial<<<numblocksPathSegmentTracing, blockSize1d>>>(
			iter,
			cur_num_paths,
			dev_intersections,
			dev_paths,
			dev_materials
		);

		// Stream compact away all of the terminated paths.
		dev_paths_end = thrust::partition(thrust::device, dev_paths, dev_paths_end, raytracing_continuing());
	}

	// Assemble this iteration and apply it to the image
	dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
	finalGather<<<numBlocksPixels, blockSize1d>>>(num_paths, dev_image, dev_paths);

	///////////////////////////////////////////////////////////////////////////

	// Send results to OpenGL buffer for rendering
	sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

	// Retrieve image from GPU
	hipMemcpy(hst_scene->state.image.data(), dev_image, pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

	checkCUDAError("pathtrace");
}