#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include "common.h"
#include "efficient.h"
#include "thrust.h"

#define blockSize 128

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernUpSweep(int n, int d, int* in, int pow2_d, int pow2_d1) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);

            if (index >= n) return;
            if (index % pow2_d1 == 0) {
                in[index + pow2_d1 - 1] += in[index + pow2_d - 1];
            }
        }

        __global__ void kernDownSweep(int n, int d, int* in, int pow2_d, int pow2_d1) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);

            if (index >= n) return;
            if (index % pow2_d1 == 0) {
                int left = in[index + pow2_d - 1];
                in[index + pow2_d - 1] = in[index + pow2_d1 - 1];
                in[index + pow2_d1 - 1] += left;
            }
        }
        
        /**
        * Helper method to calculate the distance from the nearest power of 2 greater than or equal to n
        */
        int distanceFromPowTwo(int n) {
            int pos = ceil(log2(n));
            return int(1 << pos) - n;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int* input;
            int numItems = n;
            int zerosToPad = distanceFromPowTwo(n);
            if (zerosToPad == 0) {
                hipMalloc((void**)&input, numItems * sizeof(int));
                hipMemcpy(input, idata, sizeof(int) * numItems, hipMemcpyHostToDevice);
            }
            else {
                numItems += zerosToPad;
                hipMalloc((void**)&input, numItems * sizeof(int));
                hipMemcpy(input + zerosToPad, idata, sizeof(int) * n, hipMemcpyHostToDevice);
                hipMemset(input, 0, zerosToPad * sizeof(int));
            }
            dim3 fullBlocksPerGrid((numItems + blockSize - 1) / blockSize);
            timer().startGpuTimer();
            // up sweep
            for (int d = 0; d <= ilog2ceil(numItems) - 1; ++d) {
                int pow2_d = 1 << d;
                int pow2_d1 = 1 << (d + 1);
                kernUpSweep << <fullBlocksPerGrid, blockSize >> > (numItems, d, input, pow2_d, pow2_d1);
            }
            // down sweep
            hipMemset(input + numItems - 1, 0, sizeof(int));
            for (int d = ilog2ceil(numItems) - 1; d >= 0; --d) {
                int pow2_d = 1 << d;
                int pow2_d1 = 1 << (d + 1);
                kernDownSweep << <fullBlocksPerGrid, blockSize >> > (numItems, d, input, pow2_d, pow2_d1);
            }
            timer().endGpuTimer();
            hipMemcpy(odata, input + zerosToPad, sizeof(int) * n, hipMemcpyDeviceToHost);
            hipFree(input);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            int* input; // input array
            int* output; // output array
            int* bools; // boolean array
            int* scanned; // scan result
            int zerosToPad = distanceFromPowTwo(n);
            int numItems = n + zerosToPad;
            hipMalloc((void**)&input, numItems * sizeof(int));
            hipMemcpy(input + zerosToPad, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            hipMemset(input, 0, zerosToPad * sizeof(int));
            hipMalloc((void**)&output, numItems * sizeof(int));
            hipMalloc((void**)&bools, numItems * sizeof(int));
            hipMalloc((void**)&scanned, numItems * sizeof(int));

            dim3 fullBlocksPerGrid((numItems + blockSize - 1) / blockSize);

            timer().startGpuTimer();
            StreamCompaction::Common::kernMapToBoolean << <fullBlocksPerGrid, blockSize >> > (numItems, bools, input);
            // do scan here
            hipMemcpy(scanned, bools, sizeof(int) * numItems, hipMemcpyDeviceToDevice);
            for (int d = 0; d <= ilog2ceil(numItems) - 1; ++d) {
                int pow2_d = 1 << d;
                int pow2_d1 = 1 << (d + 1);
                kernUpSweep << <fullBlocksPerGrid, blockSize >> > (numItems, d, scanned, pow2_d, pow2_d1);
            }
            // down sweep
            hipMemset(scanned + numItems - 1, 0, sizeof(int));
            for (int d = ilog2ceil(numItems) - 1; d >= 0; --d) {
                int pow2_d = 1 << d;
                int pow2_d1 = 1 << (d + 1);
                kernDownSweep << <fullBlocksPerGrid, blockSize >> > (numItems, d, scanned, pow2_d, pow2_d1);
            }
            StreamCompaction::Common::kernScatter << < fullBlocksPerGrid, blockSize >> > (numItems, output, input, bools, scanned);
            timer().endGpuTimer();
            hipMemcpy(odata, output, sizeof(int) * n, hipMemcpyDeviceToHost);
            int result = 0;
            for (int i = 0; i < n; ++i) {
                if (odata[i] == 0) continue;
                result++;
            }
            hipFree(input);
            hipFree(output);
            hipFree(bools);
            hipFree(scanned);
            return result;
        }
    }
}
